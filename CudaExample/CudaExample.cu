#include "hip/hip_runtime.h"
// Example.cpp - An example of the Particle System API in OpenGL
//
// Copyright 1999-2006, 2022 by David K. McAllister

#include "CudaExample.h"

#include "Particle/pAPI.h"
using namespace PAPI;

#include "Particle/pActionImpls.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
extern ParticleContext_t P;

Particle_t* devParticles = nullptr;
hiprandState_t* devRandState = nullptr;

__global__ void ParticleSetupKernel(hiprandState_t* devRandState, int maxParticles)
{
    int xi = blockIdx.x * blockDim.x + threadIdx.x;
    if (xi > maxParticles) return;

    devRandState[xi] = pSRandf(xi);
}

void CudaParticleSetup(int maxParticles)
{
    hipMalloc(&devParticles, sizeof(Particle_t) * maxParticles);
    hipMalloc(&devRandState, sizeof(hiprandState_t) * maxParticles);
}

__global__ void ParticleDynamics(Particle_t* devParticles, int maxParticles, float dt)
{
    int xi = blockIdx.x * blockDim.x + threadIdx.x;
    if (xi > maxParticles) return;

    Particle_t& p_ = devParticles[xi];

    // Gravity
    PAGravity_Impl(p_, dt, pVec(0.f, 0.f, -0.01f));

    // Bounce particles off a disc of radius 5
    PABounceDisc_Impl(p_, dt, PDDisc(pVec(0.f, 0.f, 0.f), pVec(0.f, 0.f, 1.f), 5.f), 0.f, 0.5f, 0.f);

    // Kill particles below Z=-3
    PASink_Impl(p_, dt, false, PDPlane(pVec(0.f, 0.f, -3.f), pVec(0.f, 0.f, 1.f)));

    // Move particles to their new positions
    PAMove_Impl(p_, dt, true, false);
}

// A water fountain spraying upward
void ComputeParticles()
{
    // Set the state of the new particles to be generated
    pSourceState S;
    S.Velocity(PDCylinder(pVec(0.0f, -0.01f, 0.25f), pVec(0.0f, -0.01f, 0.27f), 0.021f, 0.019f));
    S.Color(PDLine(pVec(0.8f, 0.9f, 1.0f), pVec(1.0f, 1.0f, 1.0f)));

    // Generate particles along a very small line in the nozzle
    P.Source(200, PDLine(pVec(0.f, 0.f, 0.f), pVec(0.f, 0.f, 0.4f)), S);

    dim3 threads(32);
    dim3 grid(P.GetGroupCount());

    ParticleDynamics<<<grid, threads>>>(devParticles, P.GetGroupCount(), P.GetTimeStep());

    P.CommitKills();
}
